#define N 325000000    
#define M 1024 
#include <iostream>
#include "hip/hip_runtime.h"




__global__ void add(float *a, float *b, float *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) c[index] = a[index] + b[index];
}



void random_floats(float* x, int count) {
    for (int i = 0; i < count; i++) 
        x[i] = (rand() % 1000 - 500) / 1000.0;
}



void check_results(float* x, float* y, float* z, int count) {
    for (int i = 0; i < count; i++) 
        if (abs(x[i] + y[i] - z[i]) > pow(10, -8))
            return;

    printf("Check completed correctly\n");
}



int main() {
    float time_transaction1 = 0;
    float time_transaction2 = 0;
    float time_calculation = 0;
    hipEvent_t start_transaction1, stop_transaction1, start_transaction2, stop_transaction2, start_calculation, stop_calculation;
    hipEventCreate(&start_transaction1);
    hipEventCreate(&stop_transaction1);
    hipEventCreate(&start_transaction2);
    hipEventCreate(&stop_transaction2);
    hipEventCreate(&start_calculation);
    hipEventCreate(&stop_calculation);



    int size = N * sizeof(float);    
    
    float *gpu_a;
    float *gpu_b;
    float *gpu_c;
    
    float *a = new float[size];
    float *b = new float[size];
    float *c = new float[size];



    hipMalloc((void**)&gpu_a, size);
    hipMalloc((void**)&gpu_b, size);
    hipMalloc((void**)&gpu_c, size);



    random_floats(a, N); 
    random_floats(b, N);



    hipEventRecord(start_transaction1, 0);
    hipMemcpy(gpu_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, size, hipMemcpyHostToDevice);
    hipEventRecord(stop_transaction1, 0);



    hipEventRecord(start_calculation, 0);
    add <<<(N + M - 1) / M, M >>> (gpu_a, gpu_b, gpu_c);
    hipEventRecord(stop_calculation, 0);



    hipEventRecord(start_transaction2, 0);
    hipMemcpy(c, gpu_c, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop_transaction2, 0);



    hipDeviceSynchronize();
    hipEventElapsedTime(&time_transaction1, start_transaction1, stop_transaction1);
    hipEventElapsedTime(&time_transaction2, start_transaction2, stop_transaction2);
    hipEventElapsedTime(&time_calculation, start_calculation, stop_calculation);



    check_results(a, b, c, N);



    printf("Elapsed time_transaction1: %.2f ms\n", time_transaction1);
    printf("Elapsed time_transaction2: %.2f ms\n", time_transaction2);
    printf("Elapsed time_calculation:  %.2f ms\n", time_calculation);



    hipEventDestroy(start_transaction1);
    hipEventDestroy(stop_transaction1);
    hipEventDestroy(start_transaction2);
    hipEventDestroy(stop_transaction2);
    hipEventDestroy(start_calculation);
    hipEventDestroy(stop_calculation);

    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    delete[] a;
    delete[] b;
    delete[] c;



    return 0;
}

